#include "hip/hip_runtime.h"

#ifndef RESAMPLE_BILINEAR
	#define RESAMPLE_BILINEAR

    #if __CUDACC_VER_MAJOR__ >= 9
        #include <hip/hip_fp16.h>
    #endif
	#include "PrGPU/KernelSupport/KernelCore.h" //includes KernelWrapper.h
	#include "PrGPU/KernelSupport/KernelMemory.h"

	#if GF_DEVICE_TARGET_DEVICE
		GF_KERNEL_FUNCTION(kRgb2Gray,
			((GF_PTR(float4))(inImg))
			((GF_PTR(float4))(destImg)),
			((int)(inPitch))
			((int)(destPitch))
			((int)(in16f))
			((unsigned int)(outWidth))
			((unsigned int)(outHeight)),
			((uint2)(inXY)(KERNEL_XY)))
		{
			float4  color, dest;


			if (inXY.x >= outWidth || inXY.y >= outHeight) return;

			color = ReadFloat4(inImg, inXY.y * inPitch + inXY.x, !!in16f);

			dest.x = 0.2126 * color.z + 0.7152 * color.y + 0.0722 * color.x;
			dest.y = dest.x;
			dest.z = dest.x;
			dest.w = color.w;

			WriteFloat4(dest, destImg, inXY.y * destPitch + inXY.x, !!in16f);
		}
	#endif

	#if __NVCC__
		void Rgb2Gray_CUDA (
			float *inBuf,
			float *destBuf,
			int inPitch,
			int destPitch,
			int	is16f,
			unsigned int width,
			unsigned int height)
		{
			dim3 blockDim (16, 16, 1);
			dim3 gridDim ( (width + blockDim.x - 1)/ blockDim.x, (height + blockDim.y - 1) / blockDim.y, 1 );		

			kRgb2Gray <<< gridDim, blockDim, 0 >>> ((float4*)inBuf, (float4*) destBuf, inPitch, destPitch, is16f, width, height );

			hipDeviceSynchronize();
		}
	#endif //GF_DEVICE_TARGET_HOST

#endif //SDK_CROSS_DISSOLVE
